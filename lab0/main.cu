#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

/***
 * letter toggler: toggle between lower case and upper case
 ***/
__global__ void SomeTransform(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // run through all the words
    for (; idx < fsize; idx+=blockDim.x*gridDim.x) {
        if (input_gpu[idx] == ' ' or input_gpu[idx] == '\n') continue;
    	if (input_gpu[idx] < 97) { // upper letter
		    input_gpu[idx] += 32;
        } else if (input_gpu[idx] >= 97) { //lower letter
            input_gpu[idx] -= 32;
        }
    }
}

int main(int argc, char **argv)
{
	// init, and check
	if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}
	FILE *fp = fopen(argv[1], "r");
	if (not fp) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// read files
	MemoryBuffer<char> text(fsize+1);
	auto text_smem = text.CreateSync(fsize);
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	// An example: transform the first 64 characters to '!'
	// Don't transform over the tail
	// And don't transform the line breaks
	SomeTransform<<<2, 2>>>(input_gpu, fsize);

	puts(text_smem.get_cpu_ro());
	return 0;
}
